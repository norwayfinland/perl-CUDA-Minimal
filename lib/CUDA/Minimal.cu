#include "Minimal.h"

#ifdef __cplusplus
extern "C" {
#endif

int myCudaFree (void * dev_ptr) {
	hipError_t err = hipFree(dev_ptr);
	return (int)err;
}

int myCudaFailed (int err) {
	return (hipError_t)err != hipSuccess;
}

const char * myCudaGetErrorString(int err) {
	return hipGetErrorString((hipError_t)err);
}

int myCudaMalloc(void ** dev_ptr_ptr, size_t data_len) {
	hipError_t err= hipMalloc(dev_ptr_ptr, data_len);
	return (int)err;
}

int myCudaMemcpy(void * dst_ptr, void * src_ptr, size_t length, enum myCudaMemcpyKind mykind) {
	enum hipMemcpyKind kind;
	switch (mykind) {
		case DeviceToHost: kind = hipMemcpyDeviceToHost; break;
		case HostToDevice: kind = hipMemcpyHostToDevice; break;
		case DeviceToDevice: kind = hipMemcpyDeviceToDevice; break;
	}
	hipError_t err = hipMemcpy(dst_ptr, src_ptr, length, kind);
	return (int)err;
}

void myCudaThreadSynchronize() {
	hipDeviceSynchronize();
}

int myCudaGetLastError() {
	hipError_t err = hipGetLastError();
	return (int) err;
}

int myCudaPeekAtLastError() {
	hipError_t err = hipPeekAtLastError();
	return (int) err;
}

#ifdef __cplusplus
}
#endif
